#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <hip/hip_runtime_api.h>
#include <math.h>

#include <GL/glew.h>
#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "bodysystem.h"

__constant__ float softeningSquared;
__constant__ double softeningSquared_fp64;

hipError_t setSofteningSquared(float softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared),
                              &softeningSq,
                              sizeof(float), 0,
                              hipMemcpyHostToDevice);
}

hipError_t setSofteningSquared(double softeningSq)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared_fp64),
                              &softeningSq,
                              sizeof(double), 0,
                              hipMemcpyHostToDevice);
}

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

template<typename T>
__device__ T rsqrt_T(T x)
{
    return rsqrt(x);
}

template<>
__device__ float rsqrt_T<float>(float x)
{
    return rsqrtf(x);
}

template<>
__device__ double rsqrt_T<double>(double x)
{
    return rsqrt(x);
}


// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i+blockDim.x*threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i,j) sharedPos[i+blockDim.x*j]

template <typename T>
__device__ T getSofteningSquared()
{
    return softeningSquared;
}
template <>
__device__ double getSofteningSquared<double>()
{
    return softeningSquared_fp64;
}

template <typename T>
struct DeviceData
{
    T *dPos[2]; // mapped host pointers
    T *dVel;
    hipEvent_t  event;
    unsigned int offset;
    unsigned int numBodies;
};


template <typename T>
__device__ typename vec3<T>::Type
bodyBodyInteraction(typename vec3<T>::Type ai,
                    typename vec4<T>::Type bi,
                    typename vec4<T>::Type bj)
{
    typename vec3<T>::Type r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    T distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
    distSqr += getSofteningSquared<T>();

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    T invDist = rsqrt_T(distSqr);
    T invDistCube =  invDist * invDist * invDist;

    // s = m_j * invDistCube [1 FLOP]
    T s = bj.w * invDistCube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

template <typename T>
__device__ typename vec3<T>::Type
computeBodyAccel(typename vec4<T>::Type bodyPos,
                 typename vec4<T>::Type *positions,
                 int numTiles)
{
    typename vec4<T>::Type *sharedPos = SharedMemory<typename vec4<T>::Type>();

    typename vec3<T>::Type acc = {0.0f, 0.0f, 0.0f};

    for (int tile = 0; tile < numTiles; tile++)
    {
        sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

        __syncthreads();

        // This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[counter]);
        }

        __syncthreads();
    }

    return acc;
}

template<typename T>
__global__ void
integrateBodies(typename vec4<T>::Type *__restrict__ newPos,
                typename vec4<T>::Type *__restrict__ oldPos,
                typename vec4<T>::Type *vel,
                unsigned int deviceOffset, unsigned int deviceNumBodies,
                float deltaTime, float damping, int numTiles)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= deviceNumBodies)
    {
        return;
    }

    typename vec4<T>::Type position = oldPos[deviceOffset + index];

    typename vec3<T>::Type accel = computeBodyAccel<T>(position,
                                                       oldPos,
                                                       numTiles);

    // acceleration = force / mass;
    // new velocity = old velocity + acceleration * deltaTime
    // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
    // (because they cancel out).  Thus here force == acceleration
    typename vec4<T>::Type velocity = vel[deviceOffset + index];

    velocity.x += accel.x * deltaTime;
    velocity.y += accel.y * deltaTime;
    velocity.z += accel.z * deltaTime;

    velocity.x *= damping;
    velocity.y *= damping;
    velocity.z *= damping;

    // new position = old position + velocity * deltaTime
    position.x += velocity.x * deltaTime;
    position.y += velocity.y * deltaTime;
    position.z += velocity.z * deltaTime;

    // store new position and velocity
    newPos[deviceOffset + index] = position;
    vel[deviceOffset + index]    = velocity;
}


/* ----------------------------------------------- half the work --------------------------------------------*/


template<typename T>
__device__ void
CalculateForces(typename vec3<T>::Type index, typename vec4<T>::Type *__restrict__ oldPos, int numTiles, unsigned int deviceOffset, unsigned int deviceNumBodies, float deltaTime, float damping, typename vec4<T>::Type *vel)
{

  // work on the tile
  for (int tile_i = 0; tile_i < numTiles; tile_i++)
  {
    int index_i = (index.x - tile_i - index.y)*deviceNumBodies + (index.y  + tile_i);
    typename vec4<T>::Type pos_i = oldPos[index_i];
    
    for (int tile_j = tile_i+1; tile_j < numTiles; tile_j++) 
    {
      int index_j = (index.x - tile_j - index.y)*deviceNumBodies + (index.y  + tile_j);
      typename vec4<T>::Type pos_j = oldPos[index_j];

      typename vec3<T>::Type accel = {0.0f, 0.0f, 0.0f};

      //aceleration, body 1 , body 2
      accel = bodyBodyInteraction<T>(accel, pos_i, pos_j);

      // acceleration = force / mass;
      // new velocity = old velocity + acceleration * deltaTime
      // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
      // (because they cancel out).  Thus here force == acceleration
      typename vec4<T>::Type velocity = vel[deviceOffset + index_i];
    
      velocity.x += accel.x * deltaTime;
      velocity.y += accel.y * deltaTime;
      velocity.z += accel.z * deltaTime;

      velocity.x *= damping;
      velocity.y *= damping;
      velocity.z *= damping;

      // new position = old position + velocity * deltaTime
      pos_i.x += velocity.x * deltaTime;
      pos_i.y += velocity.y * deltaTime;
      pos_i.z += velocity.z * deltaTime;
    
      // we also update the second particle with the inverse force (velocity) by the principle of newtons third law
      pos_j.x += -velocity.x * deltaTime;
      pos_j.y += -velocity.y * deltaTime;
      pos_j.z += -velocity.z * deltaTime;
    }
    
  }
  
}

// global variable   -- not the best place for it to be --
__device__ int diagonalCounter_i;
__device__ int block_atomic;  
__device__ bool stillWorking;
__device__ int actualNumBlocks;
/* Integrate the bodies by doing only half the works thanks to newtons third law : by Herbstrith */
template<typename T>
__global__ void
integrateBodiesHalfWork(typename vec4<T>::Type *__restrict__ newPos,
                        typename vec4<T>::Type *__restrict__ oldPos,
                        typename vec4<T>::Type *vel,
                        unsigned int deviceOffset, unsigned int deviceNumBodies,
                        float deltaTime, float damping, int numTiles,
                        int blockSize, int numBlocks)
{
  // will be shared between the threads
  stillWorking = true;
  block_atomic = 0;
  actualNumBlocks = numBlocks;
  diagonalCounter_i = deviceNumBodies;
  __shared__ int diagonalCounter_j;
  diagonalCounter_j = 0;
  __shared__ bool blockEnd;
  blockEnd = false;
  
  int j_end = blockSize * blockIdx.x + blockSize + (numBlocks*blockSize*diagonalCounter_j);
  j_end = ( j_end > deviceNumBodies) ? deviceNumBodies : j_end;
  
  // x = i start, y = i end, z = j start, w = j end
  __shared__ typename vec4<T>::Type workRange;
     
  workRange.x = deviceNumBodies - diagonalCounter_i;
  workRange.y = deviceNumBodies -diagonalCounter_i;    
  workRange.z =  (blockSize * blockIdx.x) +(numBlocks*blockSize*diagonalCounter_j) ;
  workRange.w = j_end;
  
  
  while ( stillWorking ) {
    
    typename vec3<T>::Type index;
    index.x = (workRange.x - numTiles) - threadIdx.x;
    index.y = (workRange.z + numTiles) + threadIdx.x;
   
    if(index.x > 0 || index.y < deviceNumBodies) {
      // calculate  the index particle tile  on the system... each thread will work on numTiles particles
      CalculateForces<T>(index,oldPos, numTiles, deviceOffset, deviceNumBodies, deltaTime, damping, vel);
    } 
    

    //sync threads
    __syncthreads();
    
    if (threadIdx.x  == 0) {
            
     diagonalCounter_j++;            
     j_end = blockSize * blockIdx.x + blockSize + (numBlocks*blockSize*diagonalCounter_j);
     j_end = (j_end > deviceNumBodies) ? deviceNumBodies - diagonalCounter_i : j_end;
     
     workRange.x = deviceNumBodies - diagonalCounter_i;
     workRange.y = deviceNumBodies -diagonalCounter_i;
     workRange.z =  (blockSize * blockIdx.x) +(numBlocks*blockSize*diagonalCounter_j) ;
     workRange.w = j_end;

     //  we reached the end of the diagonal line
     if(workRange.z >= (deviceNumBodies - diagonalCounter_i)) {
       diagonalCounter_j = 0;
       atomicAdd(&block_atomic, 1);
       //busy waiting
       while (block_atomic < actualNumBlocks) continue;

       if (blockIdx.x == 0) {
         atomicSub(&diagonalCounter_i, 1);
         if(diagonalCounter_i < 0) {
           atomicSub(&actualNumBlocks, 1);
           stillWorking = false;
         }
         block_atomic = 0;
       }
            
       int j_end = blockSize * blockIdx.x + blockSize + (numBlocks*blockSize*diagonalCounter_j);
       j_end = (j_end > deviceNumBodies) ? deviceNumBodies - diagonalCounter_i : j_end;
     
       workRange.x = deviceNumBodies - diagonalCounter_i;
       workRange.y = deviceNumBodies -diagonalCounter_i;    
       workRange.z =  (blockSize * blockIdx.x) +(numBlocks*blockSize*diagonalCounter_j) ;
       workRange.w = j_end;
       
       // this block wont work anymore
       if(workRange.z >= (deviceNumBodies - diagonalCounter_i) && blockIdx.x != 0) {
         atomicSub(&actualNumBlocks, 1);
         blockEnd = true;
       }
       
     }
       
    }
    
    __syncthreads();

    //end the idle block threads
    if (blockEnd) {
      return;
    }

  }

}



/* ----------------------------------------------- half the work end --------------------------------------------*/



template <typename T>
void integrateNbodySystem(DeviceData<T> *deviceData,
                          hipGraphicsResource **pgres,
                          unsigned int currentRead,
                          float deltaTime,
                          float damping,
                          unsigned int numBodies,
                          unsigned int numDevices,
                          int blockSize,
                          bool bUsePBO)
{
    if (bUsePBO)
    {
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[currentRead], cudaGraphicsMapFlagsReadOnly));
        checkCudaErrors(cudaGraphicsResourceSetMapFlags(pgres[1-currentRead], cudaGraphicsMapFlagsWriteDiscard));
        checkCudaErrors(hipGraphicsMapResources(2, pgres, 0));
        size_t bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[currentRead]), &bytes, pgres[currentRead]));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&(deviceData[0].dPos[1-currentRead]), &bytes, pgres[1-currentRead]));
    }

    for (unsigned int dev = 0; dev != numDevices; dev++)
    {
        if (numDevices > 1)
        {
            hipSetDevice(dev);
        }

        int numBlocks = (deviceData[dev].numBodies + blockSize-1) / blockSize;
        int numTiles = (numBodies + blockSize - 1) / blockSize;
        numTiles = 10;
        int sharedMemSize = blockSize * 4 * sizeof(T); // 4 floats for pos

        integrateBodiesHalfWork<T><<< numBlocks, blockSize, sharedMemSize >>>
            ((typename vec4<T>::Type *)deviceData[dev].dPos[1-currentRead],
             (typename vec4<T>::Type *)deviceData[dev].dPos[currentRead],
             (typename vec4<T>::Type *)deviceData[dev].dVel,
             deviceData[dev].offset, deviceData[dev].numBodies,
             deltaTime, damping, numTiles,
             blockSize, numBlocks);
        
        if (numDevices > 1)
        {
            checkCudaErrors(hipEventRecord(deviceData[dev].event));
            // MJH: Hack on older driver versions to force kernel launches to flush!
            hipStreamQuery(0);
        }

        // check if kernel invocation generated an error
        getLastCudaError("Kernel execution failed");
    }

    if (numDevices > 1)
    {
        for (unsigned int dev = 0; dev < numDevices; dev++)
        {
            checkCudaErrors(hipEventSynchronize(deviceData[dev].event));
        }
    }

    if (bUsePBO)
    {
        checkCudaErrors(hipGraphicsUnmapResources(2, pgres, 0));
    }
}


// Explicit specializations needed to generate code
template void integrateNbodySystem<float>(DeviceData<float> *deviceData,
                                          hipGraphicsResource **pgres,
                                          unsigned int currentRead,
                                          float deltaTime,
                                          float damping,
                                          unsigned int numBodies,
                                          unsigned int numDevices,
                                          int blockSize,
                                          bool bUsePBO);

template void integrateNbodySystem<double>(DeviceData<double> *deviceData,
                                           hipGraphicsResource **pgres,
                                           unsigned int currentRead,
                                           float deltaTime,
                                           float damping,
                                           unsigned int numBodies,
                                           unsigned int numDevices,
                                           int blockSize,
                                           bool bUsePBO);
